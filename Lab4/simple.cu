#include "hip/hip_runtime.h"
// Simple CUDA example by Ingemar Ragnemalm 2009. Simplest possible?
// Assigns every element in an array with its index.

// nvcc simple.cu -L /usr/local/cuda/lib -lcudart -o simple

#include <stdio.h>
#include <math.h>

const int N = 16; 
const int blocksize = 16; 

__global__ 
void simple(float *c) 
{
	//c[threadIdx.x] = threadIdx.x;
	c[threadIdx.x] = std::sqrt(c[threadIdx.x]);

}

int main()
{
	float *c = new float[N];	
	float *cd;
	const int size = N*sizeof(float);

	for (int i = 1; i <= N; ++i) {
	    c[i - 1] = i  ;
	}   

	hipMalloc( (void**)&cd, size );
	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	hipMemcpy(cd, c, size, hipMemcpyHostToDevice); 
	simple<<<dimGrid, dimBlock>>>(cd);
	hipDeviceSynchronize();
	hipMemcpy( c, cd, size, hipMemcpyDeviceToHost ); 
	hipFree( cd );
	
	for (int i = 0; i < N; i++)
		printf("%.9f %.9f\n", c[i], std::sqrt(i + 1));
	printf("\n");
	delete[] c;
	printf("done\n");
	return EXIT_SUCCESS;
}

// 1a
// 16

// 1b
// no, because the cpu has higher float precision calculations
